#include "hip/hip_runtime.h"
#include <optix.h>

#include "Hello.h"

extern "C" 
{
    __constant__ Params params;
}

static __forceinline__ __device__ void setPayload(float3 p)
{
	optixSetPayload_0(__float_as_uint(p.x));
	optixSetPayload_1(__float_as_uint(p.y));
	optixSetPayload_2(__float_as_uint(p.z));
}

extern "C" __global__ void __raygen__rg()
{
    uint3 launch_index = optixGetLaunchIndex();
    RayGenData* rtData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());

    auto u = double(launch_index.x) / (params.image_width - 1);
    auto v = double(launch_index.y) / (params.image_height - 1);

    float3 origin = rtData->origin;
    float3 direction = rtData->lower_left_corner + u * rtData->horizontal + v * rtData->vertical - rtData->origin;

	// Trace the ray against our scene hierarchy
	unsigned int p0, p1, p2;
	optixTrace(
		params.handle,
		origin,
		direction,
		0.0f,                // Min intersection distance
		1e16f,               // Max intersection distance
		0.0f,                // rayTime -- used for motion blur
		OptixVisibilityMask(255), // Specify always visible
		OPTIX_RAY_FLAG_NONE,
		0,                   // SBT offset   -- See SBT discussion
		1,                   // SBT stride   -- See SBT discussion
		0,                   // missSBTIndex -- See SBT discussion
		p0, p1, p2);

	float3 pixel_color = make_float3(
		__uint_as_float(p0),
		__uint_as_float(p1),
		__uint_as_float(p2)
	);

    params.image[launch_index.y * params.image_width + launch_index.x] = make_float4(pixel_color, 1.0f);
}

extern "C" __global__ void __miss__ray_color()
{
    float3 unit_direction = normalize(optixGetWorldRayDirection());
    auto t = 0.5 * (unit_direction.y + 1.0);
    setPayload(lerp(make_float3(1.0), make_float3(0.5, 0.7, 1.0), t));
}